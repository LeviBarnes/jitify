
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
__global__ void K_test(float ** fields, float const * scalars, size_t n);

template <typename T>
bool are_close(T in, T out) {
  return fabs(in - out) <= 1e-5f * fabs(in);
}
#define CUDA_CHECK(A) err=hipGetLastError();if(err)std::cout<<"CUDA error on line "<<A<<". "<<hipGetLastError()<<std::endl;
int main(void) {

  typedef float2 T;
  hipError_t err;
  // Allocate
  size_t n = 128*1024;
  size_t n_fields = 32;
  size_t n_scalars = 32;
  T *d_A, *d_B;
  T *d_C;
  hipMalloc((void**)&d_A, n*sizeof(T));
  hipMalloc((void**)&d_B, n*sizeof(T));
  hipMalloc((void**)&d_C, n*sizeof(T));
  float* h_A = (float*)malloc(sizeof(T) * n);
  float* h_B = (float*)malloc(sizeof(T) * n);
  float* h_C = (float*)malloc(sizeof(T) * n);
  // Initialize data
  for (size_t q=0; q<2*n;q++)
  {
     h_A[q] = rand()*1.0/RAND_MAX - 0.5;
     h_B[q] = rand()*1.0/RAND_MAX - 0.5;
  }

  
  // Copy data
  hipMemcpy(d_A, h_A, n * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, n * sizeof(T), hipMemcpyHostToDevice);
  CUDA_CHECK(__LINE__);

  // Create device pointers to data
  T** d_fields;
  T* d_scalars;
  hipMalloc((void**)&d_scalars, n_scalars*sizeof(T));
  hipMalloc((void**)&d_fields, n_fields*sizeof(T*));
  T** h_fields = (T**)malloc(sizeof(T*)*n_fields);
  T* h_scalars = (T*)malloc(sizeof(T)*n_scalars);
  h_fields[0] = d_A;
  h_fields[1] = d_B;
  h_fields[2] = d_C;
  hipMemcpy(d_scalars, h_scalars, sizeof(T)*n_scalars, hipMemcpyHostToDevice);
  hipMemcpy(d_fields, h_fields, sizeof(T*)*n_fields, hipMemcpyHostToDevice);
  CUDA_CHECK(__LINE__);
  
  dim3 grid(128);
  dim3 block(32);
  
  K_test<<<grid,block>>>((float**)d_fields, (float*)d_scalars, 2);
  
  hipMemcpy(h_C, d_C, n*sizeof(T), hipMemcpyDeviceToHost);
  CUDA_CHECK(__LINE__);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_fields);
  hipFree(d_scalars);
  free(h_fields);
  free(h_scalars);
  CUDA_CHECK(__LINE__);

  std::cout << "(" << h_A[2] << ", " << h_A[3] << ") * (" << h_B[2] << ", " << h_B[3] << ") -> " << "(" << h_C[2] << ", " << h_C[3] << ")" << std::endl;

  bool good = are_close(h_A[2] * h_B[2] - h_A[3] * h_B[3], h_C[2]) &&
              are_close(h_A[2] * h_B[3] + h_A[3] * h_B[2], h_C[3]);
  if (good) std::cout << "Works." << std::endl;
  else std::cout << "No work." << std::endl;
   
  free(h_A);
  free(h_B);
  free(h_C);
  return !good;

}
